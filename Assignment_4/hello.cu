
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void print1() {
	printf("Hello! tid = %d bid = %d\n", threadIdx.x, blockIdx.x);
}

int main() {
	printf("Hello from Host!\n");
	print1<<<2, 3>>>();
	hipDeviceSynchronize();
	return 0;
}

