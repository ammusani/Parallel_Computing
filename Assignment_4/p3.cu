// Write a CUDA program to multiply two matrices. Input: Matrix 1 size: m X n Matrix 2 size: n X p. Output: Result matrix


// Used the Error Handler function written by Dr. Rama in his Colab shared to us on google classroom


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>


#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

__managed__ int m = 5;
__managed__ int n = 5;
__managed__ int p = 5;

int GCD(int a, int b) {
	if (!b) return a;
	return GCD(b, a % b);
}

static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void matrixMult(int *a, int *b, int *mul) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if(row < m && col < p) {
		int t = row * p + col;
		mul[t] = 0;
		for (int i = 0; i < n; i++) mul[t] += a[row * n + i] * b[i * p + col];
	}
}

int main() {
	
	scanf("%d %d %d", &m, &n, &p);
	puts(" ");
	srand(time(0));

	int *a;
	int *b;
	int *mul;
	int *c_a;
	int *c_b;
	int *c_mul;

	a = (int *)malloc(m * n * sizeof(int));
	b = (int *)malloc(n * p * sizeof(int));
	mul = (int *)malloc(m * p * sizeof(int));

	HANDLE_ERROR(hipMalloc((void **)&c_a, m * n * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)&c_b, n * p * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)&c_mul, m * p * sizeof(int)));

	for (int i = 0; i < m; i++) {
		for(int j = 0; j < n; j++) {
			a[i * n + j] = rand() % 100;

		// To see the elements of a uncomment line 66, 68  and 70, if this is 65
		// printf("%d ", a[i * n + j]);
		}
		// puts(" ");
	}
	// puts(" ");
	
	for (int i = 0; i < n; i++) {
		for(int j = 0; j < p; j++) {
			b[i * p + j] = rand() % 100;

		// To see the elements of b uncomment line 77, 79 and 81, if this is 76
		// printf("%d ", b[i * p + j]);
		}
		// puts(" ");
	}
	// puts(" ");

	HANDLE_ERROR(hipMemcpy(c_a, a, m * n * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(c_b, b, n * p * sizeof(int), hipMemcpyHostToDevice));

	int gcd = GCD(m, p);

	dim3 threads(m / gcd, p / gcd);
	dim3 blocks(gcd, gcd);

	matrixMult<<<blocks, threads>>>(c_a, c_b, c_mul);
	hipDeviceSynchronize();


	HANDLE_ERROR(hipMemcpy(mul, c_mul, m * p * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < m; i++) {
		for(int j = 0; j < p; j++) printf("%d ", mul[i * p + j]);
		puts(" ");
	}
	puts(" ");

	free(a);
	free(b);
	free(mul);
	HANDLE_ERROR(hipFree(c_a));
	HANDLE_ERROR(hipFree(c_b));
	HANDLE_ERROR(hipFree(c_mul));

	return 0;

}

		
	
	

