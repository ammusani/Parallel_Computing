#include "hip/hip_runtime.h"
// Use Thrust library to sort a vector of elements.

#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/sort.h>
#include<thrust/copy.h>

int main() {
	long n = pow(10, 7);
	srand(time(0));
	
	thrust :: host_vector<long> h(n);
	for (int i = 0; i < n; i++) h[i] = rand() % (8 * n);
	thrust :: device_vector<long> d = h;
	thrust :: sort(d.begin(), d.end());
	thrust :: copy(d.begin(), d.end(), h.begin());
	
	for (int i = 0; i < n; i++) {
		printf("%ld ", h[i]);
		if (!(i % 5)) puts("");
	}
	puts("");
	return 0;
}
