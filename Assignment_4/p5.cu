// Implement BFS on CUDA.
// The graph is not weighted but it is directed. The BFS algorithms is same just have to change the graph to unwieghted.

// Error handler was copied from Dr. Rama's colab file shared to us on google classroom


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>


#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

__managed__ int n;

static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void BFS_Kernel(int *Va, int *Ea, int *Fa, int *Xa, int *Ca) {
	int tid = threadIdx.x;
	if (tid < n) {
		if(Fa[tid]) {
			Fa[tid] = 0;
			Xa[tid] = 1;
			
			for (int i = Va[tid]; i < Va[tid + 1]; i++) {
				int j = Ea[i];

				if(!Xa[j]) {
					Ca[j] = Ca[tid] + 1;
					Fa[j] = 1;
				}
			}
		}
	}
}

int main() {
	
	srand(time(0));	
	n = 50;
	int src = rand() % n;
	int limit = 5;		// Limit on maximum number of edges from a vertex

	printf("Number of Vertices = %d\nStarting Vertex = %d\n", n, src);

	int *Va;
	int *c_Va;

	int *Ea;
	int *c_Ea;

	int end = 0;	
	Va = (int *)malloc((n + 1) * sizeof(int));
	HANDLE_ERROR(hipMalloc((void **)&c_Va, (n + 1) * sizeof(int)));
	for (int i =0; i < n; i++) {
		Va[i] = end;
		end = end + (rand() % limit);
	}
	Va[n] = end;
	HANDLE_ERROR(hipMemcpy(c_Va, Va, (n + 1) * sizeof(int), hipMemcpyHostToDevice));

	Ea = (int *)malloc(end * sizeof(int));
	HANDLE_ERROR(hipMalloc((void **)&c_Ea, end * sizeof(int)));
	for (int i = 0; i < end; i++) {
		Ea[i] = (rand()) % n;
	}
	HANDLE_ERROR(hipMemcpy(c_Ea, Ea, end * sizeof(int), hipMemcpyHostToDevice));

	/*
	   Uncomment this to see the graph
 
	for (int i = 0; i < n; i++) printf("%d ", Va[i]);
	puts(" ");
	for (int i = 0; i < end; i++) printf("%d ", Ea[i]);
	puts(" ");
	
	/**/

	int *T;
	T = (int *)malloc(n * sizeof(n));
	

	int *c_Fa;
	HANDLE_ERROR(hipMalloc((void **)&c_Fa, n * sizeof(int)));
	memset(T, 0, n * sizeof(int));
	T[src] = 1;
	HANDLE_ERROR(hipMemcpy(c_Fa, T, n * sizeof(int), hipMemcpyHostToDevice));

	int *c_Xa;
	HANDLE_ERROR(hipMalloc((void **)&c_Xa, n * sizeof(int)));
	memset(T, 0, n * sizeof(int));
	HANDLE_ERROR(hipMemcpy(c_Xa, T, n * sizeof(int), hipMemcpyHostToDevice));

	int *c_Ca;
	HANDLE_ERROR(hipMalloc((void **)&c_Ca, n * sizeof(int)));
	memset(T, -1, n * sizeof(int));
	T[src] = 0;
	HANDLE_ERROR(hipMemcpy(c_Ca, T, n * sizeof(int), hipMemcpyHostToDevice));

	int flag = 1;

	do {
		flag = 0;
		
		BFS_Kernel <<<1, n>>> (c_Va, c_Ea, c_Fa, c_Xa, c_Ca);

		hipDeviceSynchronize();
		HANDLE_ERROR(hipMemcpy(T, c_Fa, n * sizeof(int), hipMemcpyDeviceToHost));
		
		for (int i = 0; i < n; i++) {
			if (T[i]) {
				flag = 1;
				break;
			}
		}

		
	} while(flag);

	HANDLE_ERROR(hipMemcpy(T, c_Ca, n * sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i < n; i++)
		printf("cost to reach %dth node = %d\n", i, T[i]);

	printf("\nNote: -1 means you can not reach the node from the current starting node\n");

	free(Va);
	free(Ea);
	free(T);
	hipFree(c_Va);
	hipFree(c_Ea);
	hipFree(c_Fa);
	hipFree(c_Xa);
	hipFree(c_Ca);

	return 0;

}
