// Write a CUDA program to compute the sum of two arrays. Input: Number of elements in the array. Output: Array of sums

// Used the Error Handler function written by Dr. Rama in his Colab shared to us on google classroom


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>


#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

__managed__ int n = 5;


static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void sumArrays(int *a, int *b, int *sum) {
	int tid = threadIdx.x;
	if(tid < n) {
		sum[tid] = b[tid] + a[tid];
	}
}

int main() {
	
	scanf("%d", &n);
	srand(time(0));

	int *a;
	int *b;
	int *sum;
	int *c_a;
	int *c_b;
	int *c_sum;

	a = (int *)malloc(n * sizeof(int));
	b = (int *)malloc(n * sizeof(int));
	sum = (int *)malloc(n * sizeof(int));

	HANDLE_ERROR(hipMalloc((void **)&c_a, n * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)&c_b, n * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)&c_sum, n * sizeof(int)));

	for (int i = 0; i < n; i++) {
		a[i] = rand() % 1000;
		b[i] = rand() % 1000;

		// To see the elements uncomment line 54 and 56, if this is 53
		// printf("%d %d\n", a[i], b[i]);
	}
	// puts(" ");

	HANDLE_ERROR(hipMemcpy(c_a, a, n * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(c_b, b, n * sizeof(int), hipMemcpyHostToDevice));

	sumArrays<<<1, n>>>(c_a, c_b, c_sum);
	hipDeviceSynchronize();


	HANDLE_ERROR(hipMemcpy(sum, c_sum, n * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < n; i++) {
		printf("sum[%d] = %d\n", i, sum[i]);
	}
	puts(" ");

	free(a);
	free(b);
	free(sum);
	HANDLE_ERROR(hipFree(c_a));
	HANDLE_ERROR(hipFree(c_b));
	HANDLE_ERROR(hipFree(c_sum));

	return 0;

}

		
	
	

