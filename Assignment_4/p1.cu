// Write a CUDA program to compute the sum of an array of elements. Input:Number of elements in the array. Output: Array sum

// Error handler was copied from Dr. Rama's colab file shared to us on google classroom


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>


#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

__managed__ int n = 7;
__managed__ int sum = 0;


static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void sumItUp(int *arr) {
	int tid = threadIdx.x;
	if(tid < n) {
		atomicAdd(&sum, arr[tid]);
	}
}

int main() {
	
	scanf("%d", &n);
	srand(time(0));

	int *arr;
	int *c_arr;

	arr = (int *)malloc(n * sizeof(int));
	HANDLE_ERROR(hipMalloc((void **)&c_arr, n * sizeof(int)));

	for (int i = 0; i < n; i++) {
		arr[i] = rand() % 1000;

		// To see the elements uncomment line 45 and 47, if this is 44
		// printf("%d ", arr[i]);
	}
	// puts(" ");

	HANDLE_ERROR(hipMemcpy(c_arr, arr, n * sizeof(int), hipMemcpyHostToDevice));

	sumItUp<<<1, n>>>(c_arr);
	hipDeviceSynchronize();

	printf("%d\n", sum);

	free(arr);
	HANDLE_ERROR(hipFree(c_arr));
	return 0;

}

		
	
	

