// Implement SSSP on CUDA.
// The graph is not weighted but it is directed. The BFS algorithms is same just have to change the graph to unwieghted.

// Error handler was copied from Dr. Rama's colab file shared to us on google classroom


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>


#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

__managed__ int n;

static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void SSSP_Kernel1(int *Va, int *Ea, int *Wa, int *Ma, int *Ca, int *Ua) {
	int tid = threadIdx.x;
	if (tid < n) {
		if (Ma[tid]) {
			Ma[tid] = 0;
		
			for (int i = Va[tid]; i < Va[tid + 1]; i++) {
				int j = Ea[i];
				int k = Ca[tid] + Wa[j];
				if (Ua[j] > k) Ua[j] = k;
			}
		}
	}
}

__global__ void SSSP_Kernel2 (int *Va, int *Ea, int *Wa, int *Ma, int *Ca, int *Ua) {
	
	int tid = threadIdx.x;
	if (tid < n) {
		if(Ca[tid] > Ua[tid]) {
			Ca[tid] = Ua[tid];
			Ma[tid] = 1;
		}
		Ua[tid] = Ca[tid];
	}
}

int main() {
	
	srand(time(0));	
	n = 50;
	int src = rand() % n;
	int limit1 = 5;		// Limit on maximum number of edges from a vertex
	int limit2 = 500;	// Limit on weights

	printf("Number of Vertices = %d\nStarting Vertex = %d\n", n, src);

	int *Va;
	int *c_Va;

	int *Ea;
	int *c_Ea;

	int *Wa;
	int *c_Wa;

	int end = 0;	
	Va = (int *)malloc((n + 1) * sizeof(int));
	HANDLE_ERROR(hipMalloc((void **)&c_Va, (n + 1) * sizeof(int)));
	for (int i =0; i < n; i++) {
		Va[i] = end;
		end = end + (rand() % limit1);
	}
	Va[n] = end;
	HANDLE_ERROR(hipMemcpy(c_Va, Va, (n + 1) * sizeof(int), hipMemcpyHostToDevice));

	Ea = (int *)malloc(end * sizeof(int));
	Wa = (int *)malloc(end * sizeof(int));
	int tWeight = 0;
	HANDLE_ERROR(hipMalloc((void **)&c_Ea, end * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)&c_Wa, end * sizeof(int)));
	for (int i = 0; i < end; i++) {
		Ea[i] = (rand()) % n;
		Wa[i] = (rand()) % limit2;
		tWeight = tWeight + Wa[i];
	}
	tWeight = tWeight * 10;
	HANDLE_ERROR(hipMemcpy(c_Ea, Ea, end * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(c_Wa, Wa, end * sizeof(int), hipMemcpyHostToDevice));

	/*
	   Uncomment this to see the graph
	
	for (int i = 0; i < n; i++) printf("%d ", Va[i]);
	puts(" ");
	for (int i = 0; i < end; i++) printf("%d ", Ea[i]);
	puts(" ");
	for (int i = 0; i < end; i++) printf("%d ", Wa[i]);
	puts(" ");	
	/**/

	int *T;
	T = (int *) malloc(n * sizeof(n));
	

	int *c_Ma;
	HANDLE_ERROR(hipMalloc((void **)&c_Ma, n * sizeof(int)));
	memset(T, 0, n * sizeof(int));
	T[src] = 1;
	HANDLE_ERROR(hipMemcpy(c_Ma, T, n * sizeof(int), hipMemcpyHostToDevice));

	int *c_Ua;
	HANDLE_ERROR(hipMalloc((void **)&c_Ua, n * sizeof(int)));
	for (int i = 0; i < n; i++) T[i] = tWeight;
	T[src] = 0;
	HANDLE_ERROR(hipMemcpy(c_Ua, T, n * sizeof(int), hipMemcpyHostToDevice));

	int *c_Ca;
	HANDLE_ERROR(hipMalloc((void **)&c_Ca, n * sizeof(int)));
	HANDLE_ERROR(hipMemcpy(c_Ca, T, n * sizeof(int), hipMemcpyHostToDevice));

	int flag = 1;

	do {
		flag = 0;
		
		SSSP_Kernel1 <<<1, n>>> (c_Va, c_Ea, c_Wa, c_Ma, c_Ca, c_Ua);
		hipDeviceSynchronize();

		SSSP_Kernel2 <<<1, n>>> (c_Va, c_Ea, c_Wa, c_Ma, c_Ca, c_Ua);
		hipDeviceSynchronize();

		HANDLE_ERROR(hipMemcpy(T, c_Ma, n * sizeof(int), hipMemcpyDeviceToHost));
		
		for (int i = 0; i < n; i++) {
			if (T[i]) {
				flag = 1;
				break;
			}
		}

		
	} while(flag);

	HANDLE_ERROR(hipMemcpy(T, c_Ca, n * sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i < n; i++)
		printf("cost to reach %dth node = %d\n", i, T[i]);

	printf("\nNote: Cost = %d, means you can not reach the node from the current starting node\n", tWeight);

	free(Va);
	free(Ea);
	free(Wa);
	free(T);
	hipFree(c_Va);
	hipFree(c_Ea);
	hipFree(c_Wa);
	hipFree(c_Ma);
	hipFree(c_Ua);
	hipFree(c_Ca);

	return 0;

}
